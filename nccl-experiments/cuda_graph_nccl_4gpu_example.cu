#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <chrono>

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                         \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t res = cmd;                           \
  if (res != ncclSuccess) {                         \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(res)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

// CUDA kernel to multiply each element by 0.26
__global__ void multiplyByPointTwoSix(float* data, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    data[idx] *= 0.26f;
  }
}

void runWithoutCudaGraph(float** buff, hipStream_t* s, ncclComm_t* comms, int nDev, int size) {
  auto start = std::chrono::high_resolution_clock::now();

  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  for (int iter = 0; iter < 100; ++iter) {
    // Multiply each element in buff by 0.26
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(i));
      multiplyByPointTwoSix<<<blocksPerGrid, threadsPerBlock>>>(buff[i], size);
    }
    CUDACHECK(hipDeviceSynchronize());

    // calling NCCL communication API. Group API is required when using
    // multiple devices per thread
    NCCLCHECK(ncclGroupStart());
    for (int i = 0; i < nDev; ++i)
      NCCLCHECK(ncclAllReduce((const void*)buff[i], (void*)buff[i], size, ncclFloat, ncclSum,
          comms[i], s[i]));
    NCCLCHECK(ncclGroupEnd());

    // synchronizing on CUDA streams to wait for completion of NCCL operation
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(i));
      CUDACHECK(hipStreamSynchronize(s[i]));
    }
  }

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  printf("Time without CUDA graph: %f seconds\n", elapsed.count());
}

void runWithCudaGraph(float** buff, hipStream_t* s, ncclComm_t* comms, int nDev, int size) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  hipGraph_t graph;
  hipGraphExec_t graphExec;

  hipStream_t captureStream;
  CUDACHECK(hipStreamCreate(&captureStream));
  CUDACHECK(hipStreamBeginCapture(captureStream, hipStreamCaptureModeGlobal));

  for (int iter = 0; iter < 100; ++iter) {
    // Multiply each element in buff by 0.26
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(i));
      multiplyByPointTwoSix<<<blocksPerGrid, threadsPerBlock, 0, captureStream>>>(buff[i], size);
    }

    // calling NCCL communication API. Group API is required when using
    // multiple devices per thread
    NCCLCHECK(ncclGroupStart());
    for (int i = 0; i < nDev; ++i)
      NCCLCHECK(ncclAllReduce((const void*)buff[i], (void*)buff[i], size, ncclFloat, ncclSum,
          comms[i], captureStream));
    NCCLCHECK(ncclGroupEnd());
  }
  
  CUDACHECK(hipStreamEndCapture(captureStream, &graph));
  CUDACHECK(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

  // Time measurement with CUDA graph
  auto start = std::chrono::high_resolution_clock::now();

  CUDACHECK(hipGraphLaunch(graphExec, captureStream));
  CUDACHECK(hipStreamSynchronize(captureStream));

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  printf("Time with CUDA graph: %f seconds\n", elapsed.count());

  CUDACHECK(hipStreamDestroy(captureStream));
}

void printBufferValues(float** buff, float* hostBuff, int nDev, int size, const char* label) {
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMemcpy(hostBuff, buff[i], size * sizeof(float), hipMemcpyDeviceToHost));
    printf("%s - Buff of device %d:\n", label, i);
    for (int j = 0; j < 10; ++j) { // Print only the first 10 elements for brevity
      printf("%f ", hostBuff[j]);
    }
    printf("\n");
  }
}

int main(int argc, char* argv[])
{
  ncclComm_t comms[4];

  // managing 4 devices
  int nDev = 4;
  int size = 32*1024*1024;
  int devs[4] = { 0, 1, 2, 3 };

  // allocating and initializing device buffers
  float** buff = (float**)malloc(nDev * sizeof(float*));
  float* hostBuff = (float*)malloc(size * sizeof(float));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

  // Initialize hostBuff with 1.0f
  for (int i = 0; i < size; ++i) {
    hostBuff[i] = 1.0f;
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc((void**)&buff[i], size * sizeof(float)));
    CUDACHECK(hipMemcpy(buff[i], hostBuff, size * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipStreamCreate(s+i));
  }

  // initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  // Run without CUDA graph
  runWithoutCudaGraph(buff, s, comms, nDev, size);
  printBufferValues(buff, hostBuff, nDev, size, "Without CUDA graph");

  // Reset buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMemcpy(buff[i], hostBuff, size * sizeof(float), hipMemcpyHostToDevice));
  }

  // Run with CUDA graph
  runWithCudaGraph(buff, s, comms, nDev, size);
  printBufferValues(buff, hostBuff, nDev, size, "With CUDA graph");

  // Free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(buff[i]));
  }

  free(buff);
  free(hostBuff);
  free(s);

  // Finalizing NCCL
  for (int i = 0; i < nDev; ++i)
    ncclCommDestroy(comms[i]);

  printf("Success \n");
  return 0;
}
