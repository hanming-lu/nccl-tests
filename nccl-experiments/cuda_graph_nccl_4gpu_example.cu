#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <chrono>

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                         \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t res = cmd;                           \
  if (res != ncclSuccess) {                         \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(res)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

// CUDA kernel to multiply each element by 0.26
__global__ void multiplyByPointTwoSix(float* data, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    data[idx] *= 0.26f;
  }
}

int main(int argc, char* argv[])
{
  ncclComm_t comms[4];

  // managing 4 devices
  int nDev = 4;
  int size = 32*1024*1024;
  int devs[4] = { 0, 1, 2, 3 };

  // allocating and initializing device buffers
  float** buff = (float**)malloc(nDev * sizeof(float*));
  float* hostBuff = (float*)malloc(size * sizeof(float));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

  // Initialize hostBuff with 1.0f
  for (int i = 0; i < size; ++i) {
    hostBuff[i] = 1.0f;
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc((void**)&buff[i], size * sizeof(float)));
    CUDACHECK(hipMemcpy(buff[i], hostBuff, size * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipStreamCreate(s+i));
  }

  // initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  // Time measurement without CUDA graph
  auto start = std::chrono::high_resolution_clock::now();

  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  for (int iter = 0; iter < 100; ++iter) {
    // Multiply each element in buff by 0.26
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(i));
      multiplyByPointTwoSix<<<blocksPerGrid, threadsPerBlock>>>(buff[i], size);
      CUDACHECK(hipDeviceSynchronize());
    }

    // calling NCCL communication API. Group API is required when using
    // multiple devices per thread
    NCCLCHECK(ncclGroupStart());
    for (int i = 0; i < nDev; ++i)
      NCCLCHECK(ncclAllReduce((const void*)buff[i], (void*)buff[i], size, ncclFloat, ncclSum,
          comms[i], s[i]));
    NCCLCHECK(ncclGroupEnd());

    // synchronizing on CUDA streams to wait for completion of NCCL operation
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(i));
      CUDACHECK(hipStreamSynchronize(s[i]));
    }
  }

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  printf("Time without CUDA graph: %f seconds\n", elapsed.count());

  // Print elements of buff
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMemcpy(hostBuff, buff[i], size * sizeof(float), hipMemcpyDeviceToHost));
    printf("Buff of device %d:\n", i);
    for (int j = 0; j < 10; ++j) { // Print only the first 10 elements for brevity
      printf("%f ", hostBuff[j]);
    }
    printf("\n");
  }

  
  // Reset hostBuff with 1.0f
  for (int i = 0; i < size; ++i) {
    hostBuff[i] = 1.0f;
  }
  // Reset buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMemcpy(buff[i], hostBuff, size * sizeof(float), hipMemcpyHostToDevice));
  }

  // CUDA graph setup
  hipGraph_t* graphs = (hipGraph_t*)malloc(nDev * sizeof(hipGraph_t));
  hipGraphExec_t* graphExecs = (hipGraphExec_t*)malloc(nDev * sizeof(hipGraphExec_t));

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamBeginCapture(s[i], hipStreamCaptureModeGlobal));
    for (int iter = 0; iter < 100; ++iter) {
      multiplyByPointTwoSix<<<blocksPerGrid, threadsPerBlock, 0, s[i]>>>(buff[i], size);
      // CUDACHECK(cudaStreamSynchronize(s[i]));

      NCCLCHECK(ncclGroupStart());
      NCCLCHECK(ncclAllReduce((const void*)buff[i], (void*)buff[i], size, ncclFloat, ncclSum, comms[i], s[i]));
      NCCLCHECK(ncclGroupEnd());
    }
    CUDACHECK(hipStreamEndCapture(s[i], &graphs[i]));
    CUDACHECK(hipGraphInstantiate(&graphExecs[i], graphs[i], NULL, NULL, 0));
  }

  // Time measurement with CUDA graph
  start = std::chrono::high_resolution_clock::now();

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipGraphLaunch(graphExecs[i], s[i]));
  }
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  end = std::chrono::high_resolution_clock::now();
  elapsed = end - start;
  printf("Time with CUDA graph: %f seconds\n", elapsed.count());

  // Print elements of buff
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMemcpy(hostBuff, buff[i], size * sizeof(float), hipMemcpyDeviceToHost));
    printf("Buff of device %d:\n", i);
    for (int j = 0; j < 10; ++j) { // Print only the first 10 elements for brevity
      printf("%f ", hostBuff[j]);
    }
    printf("\n");
  }

  // Free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(buff[i]));
    CUDACHECK(hipGraphDestroy(graphs[i]));
    CUDACHECK(hipGraphExecDestroy(graphExecs[i]));
  }

  free(buff);
  free(hostBuff);
  free(s);
  free(graphs);
  free(graphExecs);

  // Finalizing NCCL
  for (int i = 0; i < nDev; ++i)
    ncclCommDestroy(comms[i]);

  printf("Success \n");
  return 0;
}
