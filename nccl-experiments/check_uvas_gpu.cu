#include <iostream>
#include <hip/hip_runtime.h>

void checkP2PandUVAS() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "Error getting device count: " << hipGetErrorString(err) << std::endl;
        return;
    }

    std::cout << "Number of GPUs: " << deviceCount << std::endl;

    bool allSupportUVAS = true;

    for (int i = 0; i < deviceCount; ++i) {
        hipSetDevice(i);
        hipDeviceProp_t deviceProp;
        err = hipGetDeviceProperties(&deviceProp, i);
        if (err != hipSuccess) {
            std::cerr << "Error getting device properties for device " << i << ": " << hipGetErrorString(err) << std::endl;
            allSupportUVAS = false;
            continue;
        }

        std::cout << "Device " << i << ": " << deviceProp.name << std::endl;
        std::cout << "  Unified Addressing: " << (deviceProp.unifiedAddressing ? "Yes" : "No") << std::endl;

        if (!deviceProp.unifiedAddressing) {
            allSupportUVAS = false;
        }
    }

    if (allSupportUVAS) {
        std::cout << "All GPUs support Unified Virtual Address Space (UVAS)." << std::endl;
    } else {
        std::cout << "Not all GPUs support Unified Virtual Address Space (UVAS)." << std::endl;
    }

    std::cout << "Checking Peer-to-Peer (P2P) access between GPUs..." << std::endl;

    for (int i = 0; i < deviceCount; ++i) {
        for (int j = 0; j < deviceCount; ++j) {
            if (i != j) {
                int canAccessPeer = 0;
                hipDeviceCanAccessPeer(&canAccessPeer, i, j);
                if (canAccessPeer) {
                    std::cout << "GPU " << i << " can access GPU " << j << std::endl;
                    hipSetDevice(i);
                    hipDeviceEnablePeerAccess(j, 0);
                } else {
                    std::cout << "GPU " << i << " cannot access GPU " << j << std::endl;
                }
            }
        }
    }
}

int main() {
    checkP2PandUVAS();
    return 0;
}
