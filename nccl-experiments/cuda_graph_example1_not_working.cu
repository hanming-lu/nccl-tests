#include <iostream>
#include <hip/hip_runtime.h>

// Define the vector size and number of repetitions
const int N = 1 << 20; // 1M elements
const int REPEATS = 1000; // Number of repetitions

// Kernel for vector addition
__global__ void vecAdd(float* A, float* B, float* C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}

// Kernel for vector addition multiple times
__global__ void vecAddMultipleTimes(float* A, float* B, float* C, int n, int repetitions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < repetitions; ++i) {
        if (idx < n) {
            C[idx] = A[idx] + B[idx];
        }
    }
}

// Kernel for a single vector addition operation
__global__ void vecAddSingleOperation(float* A, float* B, float* C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}

void checkCuda(hipError_t result, const char *msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << msg << " - " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}

int main() {
    // Allocate and initialize host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    checkCuda(hipMalloc(&d_A, N * sizeof(float)), "hipMalloc A");
    checkCuda(hipMalloc(&d_B, N * sizeof(float)), "hipMalloc B");
    checkCuda(hipMalloc(&d_C, N * sizeof(float)), "hipMalloc C");

    // Copy data from host to device
    checkCuda(hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy A");
    checkCuda(hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy B");

    // Define kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start), "hipEventCreate start");
    checkCuda(hipEventCreate(&stop), "hipEventCreate stop");
    checkCuda(hipEventSynchronize(stop), "hipEventSynchronize stop");
    float milliseconds = 0;

    // Measure latency with CUDA Graphs
    hipStream_t stream;
    hipGraph_t graph;
    hipGraphExec_t instance;

    checkCuda(hipStreamCreate(&stream), "hipStreamCreate");

    checkCuda(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal), "hipStreamBeginCapture");
    for (int i = 0; i < REPEATS; ++i) {
        vecAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, N);
    }
    checkCuda(hipStreamEndCapture(stream, &graph), "hipStreamEndCapture");
    checkCuda(hipGraphInstantiate(&instance, graph, nullptr, nullptr, 0), "hipGraphInstantiate");

    checkCuda(hipEventRecord(start), "hipEventRecord start");
    checkCuda(hipGraphLaunch(instance, stream), "hipGraphLaunch");
    checkCuda(hipStreamSynchronize(stream), "hipStreamSynchronize");
    checkCuda(hipEventRecord(stop), "hipEventRecord stop");

    checkCuda(hipEventElapsedTime(&milliseconds, start, stop), "hipEventElapsedTime");
    std::cout << "Time with 100 kernels, 1 operation each [CUDA Graphs]: " << milliseconds << " ms" << std::endl;
    float overheadWithGraphs = milliseconds / REPEATS;
    std::cout << "Overhead of each kernel: " << overheadWithGraphs << " ms" << std::endl;

    // Measure latency without CUDA Graphs
    checkCuda(hipEventRecord(start), "hipEventRecord start");
    for (int i = 0; i < REPEATS; ++i) {
        vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    }
    checkCuda(hipEventRecord(stop), "hipEventRecord stop");

    checkCuda(hipEventElapsedTime(&milliseconds, start, stop), "hipEventElapsedTime");
    std::cout << "Time with 100 kernels, 1 operation each [no CUDA Graphs]: " << milliseconds << " ms" << std::endl;
    float overheadWithoutGraphs = milliseconds / REPEATS;
    std::cout << "Overhead of each kernel: " << overheadWithoutGraphs << " ms" << std::endl;

    // Measure latency with single kernel doing one operation
    checkCuda(hipEventRecord(start), "hipEventRecord start");
    vecAddSingleOperation<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    checkCuda(hipEventRecord(stop), "hipEventRecord stop");
    checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize");

    checkCuda(hipEventElapsedTime(&milliseconds, start, stop), "hipEventElapsedTime");
    std::cout << "Time with 1 kernel, 1 operation each: " << milliseconds << " ms" << std::endl;

    // Measure latency with single kernel doing REPEATS operations
    checkCuda(hipEventRecord(start), "hipEventRecord start");
    vecAddMultipleTimes<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N, REPEATS);
    checkCuda(hipEventRecord(stop), "hipEventRecord stop");
    checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize");

    checkCuda(hipEventElapsedTime(&milliseconds, start, stop), "hipEventElapsedTime");
    std::cout << "Time with 1 kernel, " << REPEATS << " operations each: " << milliseconds << " ms" << std::endl;   

    // Clean up
    checkCuda(hipGraphDestroy(graph), "hipGraphDestroy");
    checkCuda(hipGraphExecDestroy(instance), "hipGraphExecDestroy");
    checkCuda(hipStreamDestroy(stream), "hipStreamDestroy");
    checkCuda(hipEventDestroy(start), "hipEventDestroy start");
    checkCuda(hipEventDestroy(stop), "hipEventDestroy stop");

    checkCuda(hipFree(d_A), "hipFree A");
    checkCuda(hipFree(d_B), "hipFree B");
    checkCuda(hipFree(d_C), "hipFree C");

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    std::cout << "Completed successfully!" << std::endl;
    return 0;
}
