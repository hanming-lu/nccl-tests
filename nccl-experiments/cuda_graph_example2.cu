#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

void checkCuda(hipError_t result, const char *msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << msg << " - " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}

// Kernel function
__global__ void kernel1(int *data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 1024 * 1024) { // Ensuring we don't go out of bounds
        data[idx] = idx;
    }
}

__global__ void kernel2(int *data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 1024 * 1024) { // Ensuring we don't go out of bounds
        data[idx] *= 2;
    }
}

// Function to measure execution time without CUDA Graphs
void withoutCudaGraphs(int *d_data, int N) {
    hipEvent_t estart, estop;
    float milliseconds = 0;
    checkCuda(hipEventCreate(&estart), "hipEventCreate start");
    checkCuda(hipEventCreate(&estop), "hipEventCreate stop");
    checkCuda(hipEventRecord(estart), "hipEventRecord start");
    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < 1000; ++i) {
        // Launch kernels
        kernel1<<<N/256, 256>>>(d_data);
        kernel2<<<N/256, 256>>>(d_data);
    }

    checkCuda(hipEventSynchronize(estop), "hipEventSynchronize stop");
    checkCuda(hipEventRecord(estop), "hipEventRecord stop");
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    
    std::chrono::duration<double> diff = end - start;
    checkCuda(hipEventElapsedTime(&milliseconds, estart, estop), "hipEventElapsedTime");

    std::cout << "Without CUDA Graphs: " << diff.count() << " s; cuda event: " << milliseconds << " ms\n";
}

// Function to measure execution time with CUDA Graphs
void withCudaGraphs(int *d_data, int N) {
    hipGraph_t graph;
    hipGraphExec_t instance;
    hipStream_t stream;
    
    hipEvent_t estart, estop;
    float milliseconds = 0;
    checkCuda(hipEventCreate(&estart), "hipEventCreate start");
    checkCuda(hipEventCreate(&estop), "hipEventCreate stop");

    hipStreamCreate(&stream);

    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    for (int i = 0; i < 1000; ++i) {
        // Launch kernels
        kernel1<<<N/256, 256, 0, stream>>>(d_data);
        kernel2<<<N/256, 256, 0, stream>>>(d_data);
    }

    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

    checkCuda(hipEventRecord(estart), "hipEventRecord start");
    auto start = std::chrono::high_resolution_clock::now();

    hipGraphLaunch(instance, stream);
    checkCuda(hipEventRecord(estop), "hipEventRecord stop");
    hipStreamSynchronize(stream);    

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    checkCuda(hipEventElapsedTime(&milliseconds, estart, estop), "hipEventElapsedTime");

    std::cout << "With CUDA Graphs: " << diff.count() << " s; cuda event: " << milliseconds << " ms\n";

    hipGraphDestroy(graph);
    hipGraphExecDestroy(instance);
    hipStreamDestroy(stream);
}

int main() {
    const int N = 1024 * 1024;
    int *d_data;

    hipMalloc(&d_data, N * sizeof(int));

    // Measure execution time without CUDA Graphs
    withoutCudaGraphs(d_data, N);

    // Measure execution time with CUDA Graphs
    withCudaGraphs(d_data, N);

    hipFree(d_data);
    return 0;
}
