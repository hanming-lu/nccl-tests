#include "hip/hip_runtime.h"
#include <iostream>
#include <nccl.h>
#include <hip/hip_runtime.h>

#define N 1024  // Size of the array

void checkNcclStatus(ncclResult_t status, const char *msg) {
    if (status != ncclSuccess) {
        std::cerr << "NCCL error: " << msg << " - " << ncclGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCudaStatus(hipError_t status, const char *msg) {
    if (status != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " - " << hipGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void foo_kernel(float *data, int n, ncclComm_t comm, hipStream_t stream) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] = 1.0f;
    }

    // Call ncclAllReduce inside the kernel
    __syncthreads();
    if (threadIdx.x == 0) {
        ncclAllReduce(data, data, n, ncclFloat, ncclSum, comm, stream);
    }
}

int main() {
    int nGPUs;
    hipGetDeviceCount(&nGPUs);

    if (nGPUs < 2) {
        std::cerr << "This example requires at least 2 GPUs" << std::endl;
        return EXIT_FAILURE;
    }

    // Allocate host memory
    float *h_data = (float*)malloc(N * sizeof(float));

    // Allocate device memory and copy data to GPUs
    float *d_data[nGPUs];
    hipStream_t streams[nGPUs];
    for (int i = 0; i < nGPUs; ++i) {
        hipSetDevice(i);
        checkCudaStatus(hipMalloc(&d_data[i], N * sizeof(float)), "hipMalloc");
        checkCudaStatus(hipStreamCreate(&streams[i]), "hipStreamCreate");
    }

    // Initialize NCCL
    ncclComm_t comms[nGPUs];
    ncclUniqueId id;
    ncclGetUniqueId(&id);
    checkNcclStatus(ncclGroupStart(), "ncclGroupStart");
    for (int i = 0; i < nGPUs; ++i) {
        hipSetDevice(i);
        checkNcclStatus(ncclCommInitRank(&comms[i], nGPUs, id, i), "ncclCommInitRank");
    }
    checkNcclStatus(ncclGroupEnd(), "ncclGroupEnd");

    // Launch the kernel
    for (int i = 0; i < nGPUs; ++i) {
        hipSetDevice(i);
        foo_kernel<<<(N + 255) / 256, 256, 0, streams[i]>>>(d_data[i], N, comms[i], streams[i]);
        checkCudaStatus(hipGetLastError(), "Kernel launch");
    }

    // Synchronize and copy the results back to host
    for (int i = 0; i < nGPUs; ++i) {
        hipSetDevice(i);
        checkCudaStatus(hipStreamSynchronize(streams[i]), "hipStreamSynchronize");
        checkCudaStatus(hipMemcpy(h_data, d_data[i], N * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy");
        // Verify the results
        for (int j = 0; j < N; ++j) {
            if (h_data[j] != (float)nGPUs) {
                std::cerr << "Data verification failed at index " << j << ": " << h_data[j] << " != " << nGPUs << std::endl;
                return EXIT_FAILURE;
            }
        }
    }

    std::cout << "NCCL AllReduce test passed." << std::endl;

    // Cleanup
    for (int i = 0; i < nGPUs; ++i) {
        hipSetDevice(i);
        checkCudaStatus(hipFree(d_data[i]), "hipFree");
        ncclCommDestroy(comms[i]);
        checkCudaStatus(hipStreamDestroy(streams[i]), "hipStreamDestroy");
    }
    free(h_data);

    return EXIT_SUCCESS;
}


// #include <iostream>
// #include <nccl.h>
// #include <hip/hip_runtime.h>

// #define N 1024  // Size of the array

// void checkNcclStatus(ncclResult_t status, const char *msg) {
//     if (status != ncclSuccess) {
//         std::cerr << "NCCL error: " << msg << " - " << ncclGetErrorString(status) << std::endl;
//         exit(EXIT_FAILURE);
//     }
// }

// void checkCudaStatus(hipError_t status, const char *msg) {
//     if (status != hipSuccess) {
//         std::cerr << "CUDA error: " << msg << " - " << hipGetErrorString(status) << std::endl;
//         exit(EXIT_FAILURE);
//     }
// }

// __global__ void foo_kernel(float *data, int n) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < n) {
//         data[idx] = 1.0f;
//     }
// }

// int main() {
//     int nGPUs;
//     hipGetDeviceCount(&nGPUs);

//     if (nGPUs < 2) {
//         std::cerr << "This example requires at least 2 GPUs" << std::endl;
//         return EXIT_FAILURE;
//     }

//     // Allocate host memory
//     float *h_data = (float*)malloc(N * sizeof(float));
//     // for (int i = 0; i < N; ++i) {
//     //     h_data[i] = 1.0f;  // Initialize the array with 1.0
//     // }

//     // Allocate device memory and copy data to GPUs
//     float *d_data[nGPUs];
//     for (int i = 0; i < nGPUs; ++i) {
//         hipSetDevice(i);
//         checkCudaStatus(hipMalloc(&d_data[i], N * sizeof(float)), "hipMalloc");
//         hipError_t err = hipSuccess;
//         foo_kernel<<<(N + 255) / 256, 256>>>(d_data[i], N);
//         err = hipGetLastError();
//         if (err != hipSuccess) {
//             fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
//             return -1;
//           }
//         // checkCudaStatus(hipMemcpy(d_data[i], h_data, N * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
//     }

//     // Initialize NCCL
//     ncclComm_t comms[nGPUs];
//     ncclUniqueId id;
//     ncclGetUniqueId(&id);
//     checkNcclStatus(ncclGroupStart(), "ncclGroupStart");
//     for (int i = 0; i < nGPUs; ++i) {
//         hipSetDevice(i);
//         checkNcclStatus(ncclCommInitRank(&comms[i], nGPUs, id, i), "ncclCommInitRank");
//     }
//     checkNcclStatus(ncclGroupEnd(), "ncclGroupEnd");

//     // Perform NCCL AllReduce
//     checkNcclStatus(ncclGroupStart(), "ncclGroupStart");
//     for (int i = 0; i < nGPUs; ++i) {
//         hipSetDevice(i);
//         checkNcclStatus(ncclAllReduce(d_data[i], d_data[i], N, ncclFloat, ncclSum, comms[i], hipStreamDefault), "ncclAllReduce");
//     }
//     checkNcclStatus(ncclGroupEnd(), "ncclGroupEnd");

//     // Copy the results back to host
//     for (int i = 0; i < nGPUs; ++i) {
//         hipSetDevice(i);
//         checkCudaStatus(hipMemcpy(h_data, d_data[i], N * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy");
//         // Verify the results
//         for (int j = 0; j < N; ++j) {
//             if (h_data[j] != (float)nGPUs) {
//                 std::cerr << "Data verification failed at index " << j << ": " << h_data[j] << " != " << nGPUs << std::endl;
//                 return EXIT_FAILURE;
//             }
//         }
//     }

//     std::cout << "NCCL AllReduce test passed." << std::endl;

//     // Cleanup
//     for (int i = 0; i < nGPUs; ++i) {
//         hipSetDevice(i);
//         checkCudaStatus(hipFree(d_data[i]), "hipFree");
//         ncclCommDestroy(comms[i]);
//     }
//     free(h_data);

//     return EXIT_SUCCESS;
// }
